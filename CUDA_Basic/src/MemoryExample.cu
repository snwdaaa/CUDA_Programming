﻿#include "hip/hip_runtime.h"

#include <stdio.h>

void checkDeviceMemory(void) {
    size_t free, total;
    hipMemGetInfo(&free, &total);
    printf("Device memory (free/total) = %lld/%lld bytes\n", free, total);
}

int main(void) {
    int* dDataPtr;
    hipError_t errorCode;

    // 디바이스 메모리 할당
    checkDeviceMemory();
    errorCode = hipMalloc(&dDataPtr, sizeof(int) * 1024 * 1024);
    printf("hipMalloc - %s\n", hipGetErrorName(errorCode));
    checkDeviceMemory();

    // 메모리 초기화
    errorCode = hipMemset(dDataPtr, 0, sizeof(int) * 1024 * 1024);
    printf("hipMemset - %s\n", hipGetErrorName(errorCode));

    errorCode = hipFree(dDataPtr);
    printf("hipFree - %s\n", hipGetErrorName(errorCode));
    checkDeviceMemory();
}