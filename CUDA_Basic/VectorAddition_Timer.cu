#include "hip/hip_runtime.h"
// ���� ���� �ֿ� ����
// 1. Ŀ�� ���� �ð�
// Ŀ�� ȣ�� ���� �ð� ���� ������ Ŀ�� ���� ����� ������ ��� �� ���� ����
// Ŀ�� ȣ�� �� ����̽����� ��� ���� �� �ٷ� ȣ��Ʈ�� ����� ���� �� ����
// ����̽��� ���� ���� �۾� ���� ������ ��ٸ��� CUDA ����ȭ �Լ� -> hipDeviceSynchronize()
// 2. ������ ���� �ð�
// ������ ���� ���� ���� �Ŀ� ���� ���� �� ���� �ϸ� ��
// hipMemcpy() �Լ��� ȣ��Ʈ �ڵ�� ���������� ���� -> ���� ���� ������ ȣ��Ʈ ���

#include "hip/hip_runtime.h"
#include ""

#include "DS_timer.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// The size of the vector
#define NUM_DATA 1024

// Simple vector sum kernel (Max vector size : 1024)
__global__ void vecAdd(int* _a, int* _b, int* _c) {
    int tID = threadIdx.x;
    _c[tID] = _a[tID] + _b[tID];
}

int main(void)
{
    // Set Timer
    DS_timer timer(5);
    timer.setTimerName(0, (char*)"CUDA Total");
    timer.setTimerName(1, (char*)"Computation(Kernel)");
    timer.setTimerName(2, (char*)"Data Trans. : Host -> Device");
    timer.setTimerName(3, (char*)"Data Trans. : Device -> Host");
    timer.setTimerName(4, (char*)"VecAdd on Host");
    timer.initTimers();

    int* a, * b, * c, * h_c;	// Vectors on the host
    int* d_a, * d_b, * d_c;	// Vectors on the device

    int memSize = sizeof(int) * NUM_DATA;
    printf("%d elements, memSize = %d bytes\n", NUM_DATA, memSize);

    // Memory allocation on the host-side
    a = new int[NUM_DATA]; memset(a, 0, memSize);
    b = new int[NUM_DATA]; memset(b, 0, memSize);
    c = new int[NUM_DATA]; memset(c, 0, memSize);
    h_c = new int[NUM_DATA]; memset(h_c, 0, memSize);

    // Data generation
    for (int i = 0; i < NUM_DATA; i++) {
        a[i] = rand() % 10;
        b[i] = rand() % 10;
    }

    // Vector sum on host (for performance comparision)
    timer.onTimer(4);
    for (int i = 0; i < NUM_DATA; i++)
        h_c[i] = a[i] + b[i];
    timer.offTimer(4);

    //****************************************//
    //******* Write your code - start ********//

    // 1. Memory allocation on the device-side (d_a, d_b, d_c)
    hipMalloc(&d_a, memSize); hipMemset(d_a, 0, memSize);
    hipMalloc(&d_b, memSize); hipMemset(d_b, 0, memSize);
    hipMalloc(&d_c, memSize); hipMemset(d_c, 0, memSize);

    timer.onTimer(0);

    // 2. Data copy : Host (a, b) -> Device (d_a, d_b)
    timer.onTimer(2);
    hipMemcpy(d_a, a, memSize, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, memSize, hipMemcpyHostToDevice);
    timer.offTimer(2);

    // 3. Kernel call
    timer.onTimer(1);
    vecAdd << <1, NUM_DATA >> > (d_a, d_b, d_c);
    hipDeviceSynchronize();
    timer.offTimer(1);

    // 4. Copy results : Device (d_c) -> Host (c)
    timer.onTimer(3);
    hipMemcpy(c, d_c, memSize, hipMemcpyDeviceToHost);
    timer.offTimer(3);

    timer.offTimer(0);

    // 5. Release device memory (d_a, d_b, d_c)
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    timer.printTimer();

    //******** Write your code - end *********//
    //****************************************//

    // Check results
    bool result = true;
    for (int i = 0; i < NUM_DATA; i++) {
        if (h_c[i] != c[i]) {
            printf("[%d] The result is not matched! (%d, %d)\n"
                , i, h_c[i], c[i]);
            result = false;
        }
    }

    if (result)
        printf("GPU works well!\n");

    // Release host memory
    delete[] a; delete[] b; delete[] c;

    return 0;
}